#include "hip/hip_runtime.h"
#include "rempel_heat_conduction.h"

#include "constants.h"


// define variables for measuring performance
float cpu_time = 0.0;
float gpu_time = 0.0;

int main(void)
{



	// allocate cpu memory
	float * T_cpu = new float[L];
	float * T_gpu = new float[L];
	float * x = new float[Lx];

	// apply initial conditions
	initial_conditions(T_cpu);
	initial_conditions(T_gpu);

	// initialize the grid
	initial_grid(x);

	// run CPU test
	heat_1d_cpu_solve(T_cpu, x);

	// run GPU test
	heat_1d_gpu_solve(T_gpu, x);

	// calculate rms error
	float rms = 0.0;
	for(uint l = 0; l < L; l++) {
		rms += (T_cpu[l] - T_gpu[l]) * (T_cpu[l] - T_gpu[l]);
	}
	rms = std::sqrt(rms / (float) L);
	printf("CPU-GPU RMS error = %e \n", rms);

	// print something so we know it didn't crash somewhere
	printf("All tests completed!\n");

	return 0;
}

void heat_1d_gpu_solve(float * T, float * x){

	// Set up device
	int dev = 0;
	CHECK(hipSetDevice(dev));

	// Print device and precision
//	hipDeviceProp_t prop;
//	CHECK(hipGetDeviceProperties(&prop, 0));
	//		print_device_properties(prop);

	// configure the device to have the largest possible L1 cache
	CHECK(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

	// allocate pinned host memory
	float *T_h;						// dependent variables
	float *x_h;	// independent variables
	CHECK(hipHostMalloc((float **) &T_h, L * sizeof(float)));
	CHECK(hipHostMalloc((float **) &x_h, Lx * sizeof(float)));

	// allocate device memory
	float *T_d;						// dependent variables
	float *x_d;	// independent variables
	CHECK(hipMalloc((float **) &T_d, wt * Lx * sizeof(float)));
	CHECK(hipMalloc((float **) &x_d, Lx * sizeof(float)));

	// transfer initial condition from argument to pinned host memory
	CHECK(hipMemcpy(T_h, T, Lx * sizeof(float), hipMemcpyHostToHost));
	CHECK(hipMemcpy(x_h, x, Lx * sizeof(float), hipMemcpyHostToHost));

	// transfer data from the host to the device
	CHECK(hipMemcpy(T_d, T_h, Lx * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(x_d, x_h, Lx * sizeof(float), hipMemcpyHostToDevice));

	// set the number of threads and blocks
	const uint threads = lx;
	const uint blocks = Nx;

	// set the amount of shared memory
	const uint shared = 0;

	// initialize streams
	hipStream_t k_stream, m_stream;
	hipStreamCreate(&k_stream);	// initialize computation stream
	hipStreamCreate(&m_stream);	// initialize memory stream

	// initialize timing events
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	// main time-marching loop
	for(uint ti = 0; ti < Wt; ti++){	// downsampled resolution

		for(uint tj = 0; tj < wt; tj++){	// original resolution

			// start memory transfer
			if(tj == 0 and ti > 0){
				hipStreamSynchronize(m_stream); // check if memory transfer is completed
				hipMemcpyAsync(T_h + (ti * Lx), T_d, Lx * sizeof(float), hipMemcpyDeviceToHost, m_stream);
			}

			// perform timestep
			heat_1d_device_step<<<blocks, threads, shared, k_stream>>>(T_d, x_d, tj);
			hipStreamSynchronize(k_stream);


		}

	}


	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpu_time, start, stop);
	printf("gpu t =  %f ms, R = %f\n", gpu_time, cpu_time / gpu_time);


	// copy to original argument pointers
	CHECK(hipMemcpy(T, T_h, L * sizeof(float), hipMemcpyHostToHost));

	save_results("gpu/", T, x);

}

__global__ void heat_1d_device_step(float * T, float * x, uint n){

	// Find index from threadId
	uint i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i > 0 and i < (Lx - 1)){

		// Load stencil
		float T9 = T[n * Lx + (i - 1)];
		float T0 = T[n * Lx + (i + 0)];
		float T1 = T[n * Lx + (i + 1)];

		// Load position grid
		float x9 = x[i - 1];
		float x0 = x[i + 0];
		float x1 = x[i + 1];

		// compute Laplacian
		float DDx_T0 = (T9 - 2 * T0 + T1) / ((x1 - x0) * (x0 - x9));

		// compute time-update
		float Tn = T0 + dt * (kappa * DDx_T0);

		// update global memory
		T[((n + 1) % wt) * Lx + i] = Tn;

	} else {
		T[((n + 1) % wt) * Lx + i] = 0;
	}


	return;

}

void heat_1d_cpu_solve(float * T, float * x){

	struct timeval t1, t2;
	gettimeofday(&t1, 0);

	float * T_d = new float[wt * Lx];
	memcpy(T_d, T, Lx * sizeof(float));

	// main time-marching loop
	for(uint ti = 0; ti < Wt; ti++){	// downsampled resolution

		for(uint tj = 0; tj < wt; tj++){	// original resolution

			// perform timestep
			for(uint i = 0; i < Lx; i++){

				// save downsampled data to solution
				if(tj == 0 and ti > 0) {
					T[ti * Lx + i] = T_d[tj * Lx + i];
				}

				if(i > 0 and i < Lx - 1){

					// Load stencil
					float T9 = T_d[tj * Lx + (i - 1)];
					float T0 = T_d[tj * Lx + (i + 0)];
					float T1 = T_d[tj * Lx + (i + 1)];

					// Load position grid
					float x9 = x[i - 1];
					float x0 = x[i + 0];
					float x1 = x[i + 1];;

					// compute second derivative
					float DDx_T0 = (T9 - 2 * T0 + T1) / ((x1 - x0) * (x0 - x9));

					// compute time-update
					float Tn = T0 + dt * (kappa * DDx_T0);

					// update global memory
					T_d[((tj + 1) % wt) * Lx + i] = Tn;

				} else {	// boundary condition

					T_d[((tj + 1) % wt) * Lx + i] = 0;

				}



			}

		}

	}

	gettimeofday(&t2, 0);
	cpu_time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
	printf("cpu:  %f ms\n", cpu_time);

	save_results("cpu/", T, x);

	return;

}

void initial_conditions(float * T){

	// initialize host memory
	int n = 0;
	for(int i = 0; i < Lx; i++){		// Initial condition for dependent variable
		float x = i * dx;

		// Initialize temperature as rectangle function
		if(x > 0.4f and x < 0.6f){
			T[n * Lx + i] = 10.0f;
		} else {
			T[n * Lx + i] = 1.0f;
		}


	}

}

void initial_grid(float * x){

	for(int i = 0; i < Lx; i++) x[i] = i * dx;	// initialize rectangular grid in x

}

void save_results(std::string path, float * T,  float * x){

	// open files
	FILE * meta_f = fopen(("output/" + path + "meta.dat").c_str(), "wb");
	FILE * T_f = fopen(("output/" + path + "T.dat").c_str(), "wb");
	FILE * x_f = fopen(("output/" + path + "x.dat").c_str(), "wb");

	// save state variables
	fwrite(&Lt, sizeof(uint), 1, meta_f);
	fwrite(&Lx, sizeof(uint), 1, meta_f);

	// write data
	fwrite(T, sizeof(float), L, T_f);
	fwrite(x, sizeof(float), Lx, x_f);

	// close files
	fclose(meta_f);
	fclose(T_f);
	fclose(x_f);


}
