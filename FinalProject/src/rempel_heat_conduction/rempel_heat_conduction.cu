#include "hip/hip_runtime.h"
#include "rempel_heat_conduction.h"

#include "constants.h"


// define variables for measuring performance
//float cpu_time = 0.0;
//float gpu_time = 0.0;

int main(void)
{



	// allocate cpu memory
	float * T_cpu = new float[L];
	float * T_gpu = new float[L];
	float * x = new float[Lx];

	// initialize the grid
	initial_grid(x);

	// apply initial conditions
	initial_conditions(T_cpu, x);
	initial_conditions(T_gpu, x);



	// run CPU test
	float cpu_time = 0;
	cpu_time = heat_1d_cpu_solve(T_cpu, x, false);
	printf("cpu:  %f ms\n", cpu_time);

	// run GPU test
	float gpu_time = heat_1d_gpu_solve(T_gpu, x, false);
	printf("gpu t =  %f ms, R = %f\n", gpu_time, cpu_time / gpu_time);

	// calculate rms error
	float rms = 0.0;
	for(uint l = 0; l < L; l++) {
		rms += (T_cpu[l] - T_gpu[l]) * (T_cpu[l] - T_gpu[l]);
	}
	rms = std::sqrt(rms / (float) L);
	printf("CPU-GPU RMS error = %e \n", rms);

	// print something so we know it didn't crash somewhere
	printf("All tests completed!\n");

	return 0;
}

float heat_1d_gpu_solve(float * T, float * x, bool fickian){

	// Set up device
	int dev = 0;
	CHECK(hipSetDevice(dev));

	// Print device and precision
	//	hipDeviceProp_t prop;
	//	CHECK(hipGetDeviceProperties(&prop, 0));
	//		print_device_properties(prop);

	// configure the device to have the largest possible L1 cache
	CHECK(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

	// allocate pinned host memory
	float *T_h;						// dependent variables
	float *x_h;	// independent variables
	CHECK(hipHostMalloc((float **) &T_h, L * sizeof(float)));
	CHECK(hipHostMalloc((float **) &x_h, Lx * sizeof(float)));

	// allocate device memory
	float *T_d;						// dependent variables
	float *x_d;	// independent variables
	CHECK(hipMalloc((float **) &T_d, wt * Lx * sizeof(float)));
	CHECK(hipMalloc((float **) &x_d, Lx * sizeof(float)));

	// transfer initial condition from argument to pinned host memory
	CHECK(hipMemcpy(T_h, T, Lx * sizeof(float), hipMemcpyHostToHost));
	CHECK(hipMemcpy(x_h, x, Lx * sizeof(float), hipMemcpyHostToHost));

	// transfer data from the host to the device
	CHECK(hipMemcpy(T_d, T_h, Lx * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(x_d, x_h, Lx * sizeof(float), hipMemcpyHostToDevice));

	// set the number of threads and blocks
	const uint threads = lx;
	const uint blocks = Nx;

	// set the amount of shared memory
	const uint shared = 0;

	// initialize streams
	hipStream_t k_stream, m_stream;
	hipStreamCreate(&k_stream);	// initialize computation stream
	hipStreamCreate(&m_stream);	// initialize memory stream

	// initialize timing events
	float gpu_time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	// main time-marching loop
	for(uint ti = 0; ti < Wt; ti++){	// downsampled resolution

		for(uint tj = 0; tj < wt; tj++){	// original resolution

			// start memory transfer
			if(tj == 0 and ti > 0){
				hipStreamSynchronize(m_stream); // check if memory transfer is completed
				hipMemcpyAsync(T_h + (ti * Lx), T_d, Lx * sizeof(float), hipMemcpyDeviceToHost, m_stream);
			}

			// perform timestep
			heat_1d_gpu_parabolic_step<<<blocks, threads, shared, k_stream>>>(T_d, x_d, tj);
			hipStreamSynchronize(k_stream);


		}

	}


	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpu_time, start, stop);


	// copy to original argument pointers
	CHECK(hipMemcpy(T, T_h, L * sizeof(float), hipMemcpyHostToHost));

	save_results("gpu/", T, x);

	return gpu_time;

}

__global__ void heat_1d_gpu_parabolic_step(float * T, float * x, uint n){

	// Find index from threadId
	uint i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i > 0 and i < (Lx - 1)){

		// Load stencil
		float T9 = T[n * Lx + (i - 1)];
		float T0 = T[n * Lx + (i + 0)];
		float T1 = T[n * Lx + (i + 1)];

		// Load position grid
		float x9 = x[i - 1];
		float x0 = x[i + 0];
		float x1 = x[i + 1];

		// compute Laplacian
		float DDx_T0 = (T9 - 2 * T0 + T1) / ((x1 - x0) * (x0 - x9));

		// compute time-update
		float Tn = T0 + dt_p * (T0 * T0 * sqrt(T0) * DDx_T0);

		// update global memory
		T[((n + 1) % wt) * Lx + i] = Tn;

	} else if(i == 0){
		T[((n + 1) % wt) * Lx + i] = T_left;
	} else {
		T[((n + 1) % wt) * Lx + i] = T_right;
	}


	return;

}

float heat_1d_cpu_solve(float * T, float * x, bool fickian){

	float cpu_time;
	struct timeval t1, t2;
	gettimeofday(&t1, 0);

	float * T_d = new float[wt * Lx];
	memcpy(T_d, T, Lx * sizeof(float));

	// main time-marching loop
	for(uint ti = 0; ti < Wt; ti++){	// downsampled resolution

		for(uint tj = 0; tj < wt; tj++){	// original resolution

			if(tj == 0 and ti > 0) {
				memcpy(T + (ti * Lx), T_d, Lx * sizeof(float));
			}
			heat_1d_cpu_hyperbolic_step(T, T_d, x, tj);

		}

	}

	gettimeofday(&t2, 0);
	cpu_time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;


	save_results("cpu/", T, x);

	return cpu_time;

}

void heat_1d_cpu_hyperbolic_step(float * T, float * T_d, float * x, uint n){

	// perform timestep
	for(uint i = 0; i < Lx; i++){

		if(i > 0 and i < Lx - 1){

			// Load stencil
			float T9 = T_d[n * Lx + (i - 1)];
			float T0 = T_d[n * Lx + (i + 0)];
			float T1 = T_d[n * Lx + (i + 1)];
			float Tz = T_d[((n - 1) % wt) * Lx + i];

			// Load position grid
			float x9 = x[i - 1];
			float x0 = x[i + 0];
			float x1 = x[i + 1];

			float c2 = c_h * c_h;

			// compute hyperbolic timescale
//			float tau = (T0 * T0 * sqrt(T0)) / c2;
//			float tau = (T0 * T0) / c2;
			float tau = 1/c2;

			float dx2 = ((x1 - x0) * (x0 - x9));
//			float dx2 = dx * dx;

			float dt = dt_h;
			float dt2 = dt * dt;

			// compute time-update
//			float Ta = (Tz * dx2 * (dt - 2 * tau) + 2 * (c2 * (T9 - 2 * T0 + T1) * dt2 + 2 * T0 * dx2) * tau) / (dx2 * (dt + 2 * tau));

			float Ta = (T0 * dt * dx2 + c2 * (T1 - 2 * T0 + T9) * dt2 * tau + (2 * T0 - Tz) * dx2 * tau) / (dx2 * (dt + tau));

			// update global memory
			T_d[((n + 1) % wt) * Lx + i] = Ta;

		} else if(i == 0){
			T_d[((n + 1) % wt) * Lx + i] = T_left;
		} else {
			T_d[((n + 1) % wt) * Lx + i] = T_right;
		}

		if(i > 1012){
			printf("%04d %04d %f\n", n,i, T_d[((n + 1) % wt) * Lx + i]);
		}


	}

}

void heat_1d_cpu_parabolic_step(float * T, float * T_d, float * x, uint n){

	// perform timestep
	for(uint i = 0; i < Lx; i++){

		if(i > 0 and i < Lx - 1){

			// Load stencil
			float T9 = T_d[n * Lx + (i - 1)];
			float T0 = T_d[n * Lx + (i + 0)];
			float T1 = T_d[n * Lx + (i + 1)];

			// Load position grid
			float x9 = x[i - 1];
			float x0 = x[i + 0];
			float x1 = x[i + 1];;

			// compute second derivative
			float DDx_T0 = (T9 - 2 * T0 + T1) / ((x1 - x0) * (x0 - x9));

			// compute time-update
			float Tn = T0 + dt_p * (T0 * T0 * sqrt(T0) * DDx_T0);

			// update global memory
			T_d[((n + 1) % wt) * Lx + i] = Tn;

		} else if(i == 0){
			T_d[((n + 1) % wt) * Lx + i] = T_left;
		} else {
			T_d[((n + 1) % wt) * Lx + i] = T_right;
		}



	}

}

void initial_conditions(float * T, float * x){

	// initialize host memory
	for(int n = wt - 1; n < wt + 1; n++){
		printf("%d\n",n);
		for(int i = 0; i < Lx; i++){		// Initial condition for dependent variable

			// Initialize temperature as rectangle function
//			if(x[] > 0.4f and x < 0.6f){
//				T[(n % wt) * Lx + i] = 1.0f;
//			} else {
//				T[(n % wt) * Lx + i] = 1.0f;
//			}
//
			T[(n % wt) * Lx + i] = 0.1 + 0.9 * pow(x[i],5);

//			T[(n % wt) * Lx + i] = pow(pow(0.1,3.5) + (1 - pow(0.1,3.5)) * x[i], 2 / 7);
		}
	}

}

void initial_grid(float * x){

	for(int i = 0; i < Lx; i++) x[i] = i * dx;	// initialize rectangular grid in x

}

void save_results(std::string path, float * T,  float * x){

	// open files
	FILE * meta_f = fopen(("output/" + path + "meta.dat").c_str(), "wb");
	FILE * T_f = fopen(("output/" + path + "T.dat").c_str(), "wb");
	FILE * x_f = fopen(("output/" + path + "x.dat").c_str(), "wb");

	// save state variables
	fwrite(&Lt, sizeof(uint), 1, meta_f);
	fwrite(&Lx, sizeof(uint), 1, meta_f);

	// write data
	fwrite(T, sizeof(float), L, T_f);
	fwrite(x, sizeof(float), Lx, x_f);

	// close files
	fclose(meta_f);
	fclose(T_f);
	fclose(x_f);


}
